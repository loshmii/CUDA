
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloCUDA() {
    printf("Hello CUDA from GPU! \n");
}

int main() {
    helloCUDA<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}